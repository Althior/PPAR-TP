#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>

struct results
{
	float sum;
};

#include "summation_kernel.cu"

// CPU implementation
// increasing order
float log2_series(int n)
{
	int i; 
	float res=0.0;
	float tmp=1.0;
	for (i=0; i<n; i++){
		if ((i%2) == 1) {tmp = -1.0;}
		else tmp = 1.0;

		res += tmp / (i + 1);
		}
	return res;
}
// decreasing order
float log2_series_dec(int n)
{
	int i; 
	float res=0.0;
	float tmp=1.0;
	for (i=n; i>=0; i--){
		if ((i%2) == 1) {tmp = -1.0;}
		else tmp = 1.0;

		res += tmp / (i + 1);
		}
	return res;
}

int main(int argc, char ** argv)
{
    int data_size = 1024 * 1024 * 128;

    // Run CPU version
    double start_time = getclock();
    float log2 = log2_series(data_size);
    float log2_dec = log2_series_dec(data_size);
    double end_time = getclock();
    
    printf("CPU result: %.15f\n", log2);
    printf("CPU result: %.15f (dec)\n", log2_dec);
    printf(" log(2)=    %f\n", log(2.0));
    printf(" time=%fs\n", end_time - start_time);
    
    // Parameter definition
    int threads_per_block = 4 * 32;
    int blocks_in_grid = 8;
    
    int num_threads = threads_per_block * blocks_in_grid;

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    int results_size = num_threads;
    float * data_out_cpu;
    // Allocating output data on CPU
	if ((data_out_cpu = (float*) malloc(num_threads*sizeof(float))) == NULL) {printf("erreur allocation CPU"); exit(0);}

	// Allocating output data on GPU
	int i;
	float* resGPU;
	if (hipMalloc((void**)&resGPU, sizeof(float)*num_threads) != hipSuccess){printf("erreur allocation GPU"); exit(0);}

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Execute kernel
    summation_kernel<<<blocks_in_grid, threads_per_block>>>(data_size/num_threads, resGPU);

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get results back
	if (hipMemcpy(data_out_cpu, resGPU, sizeof(float)*num_threads, hipMemcpyDeviceToHost) != hipSuccess) {printf("erreur recopie resultat vers CPU"); exit(0);}
	
	// Finish reduction
    float sum = 0.;
    for (i=num_threads-1; i>=0; i--){
		sum += data_out_cpu[i];
	}
	
    // Cleanup
    hipFree(resGPU);
	free(data_out_cpu);
    
    printf("GPU results:\n");
    printf(" Sum: %f\n", sum);
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    return 0;
}

