#include "hip/hip_runtime.h"
__global__ void init_kernel(int * domain, int domain_x)
{
	// Dummy initialization
	domain[blockIdx.y * domain_x + blockIdx.x * blockDim.x + threadIdx.x]
		= (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain,
    int domain_x, int domain_y)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y;
    
	// copier mémoire globale dans la mémoire partagée pour que les threads y accedent par la suite
	// Bloc = 8*8 threads -> 100 lectures (10*10) -> |sdata| = 100
	extern __shared__ int sdata[];
	int sdataDim = 10;
	
	// chaque thread lit sa case
	int myself = read_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y);
	
	/*
	Dans le cas d'un bloc 3x3:
	bloc: x² (carré pour optimiser lectures)
	 0 1 2
	 3 4 5
	 6 7 8
	
	sdata: (x+2)² (sdataDim=5)
     0  1  2  3  4
	 5| 6  7  8| 9
	10|11 12 13|14
	15|16 17 18|19
	20 21 22 23 24
	*/
	
	// pour avoir la composante en y
	int decY, myloc;
	decY = (threadIdx.x / blockDim.x) + 1; 
	
	// position dans sdata avec contours compris
	myloc = sdataDim*decY + 1 + (threadIdx.x % blockDim.x);
	sdata[myloc] = myself;
	
	// initialiser 4 variables par thread: haut bas gauche droite
	int haut, bas, gauche, droite;
	haut   = threadIdx.x < blocDim.x;
	bas    = threadIdx.x >= blocDim.x * (blocDim.y -1);
	gauche = (threadIdx.x % blockDim.x) == 0;
	droite = ((threadIdx.x - 1) % blockDim.x) == 0;
	
	// Lectures en bordure
	if (haut) {
		// lecture en haut
		sdata[myloc-sdataDim] = read_cell(source_domain, tx, ty, 0, -1, domain_x, domain_y);
		
		if (gauche) {
			// lecture haut-gauche
			sdata[myloc-sdataDim-1] = read_cell(source_domain, tx, ty, -1, -1, domain_x, domain_y);
		}
		if (droite) {
			// lecture haut-droite
			sdata[myloc-sdataDim+1] = read_cell(source_domain, tx, ty, 1, -1, domain_x, domain_y);
		}
	}
	if (bas) {
		// lecture en bas
		sdata[myloc+sdataDim] = read_cell(source_domain, tx, ty, 0, 1, domain_x, domain_y);
		
		if (gauche) {
			// lecture bas-gauche
			sdata[myloc+sdataDim-1] = read_cell(source_domain, tx, ty, -1, 1, domain_x, domain_y);
		}
		if (droite) {
			// lecture bas-droite
			sdata[myloc+sdataDim+1] = read_cell(source_domain, tx, ty, 1, 1, domain_x, domain_y);
		}
	}
	if (gauche) {
		// lecture à gauche
		sdata[myloc-1] = read_cell(source_domain, tx, ty, -1, 0, domain_x, domain_y);
	} 
	if (droite) {
		// lecture à droite
		sdata[myloc+1] = read_cell(source_domain, tx, ty, 1, 0, domain_x, domain_y);
	}	
	
	/*
	sdata[threadIdx.x] = read_cell(source_domain, tx, ty, 0, -1, domain_x, domain_y);
	sdata[threadIdx.x+blockDim.x] = read_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y);
	sdata[threadIdx.x+(2*blockDim.x)] = read_cell(source_domain, tx, ty, 0, 1, domain_x, domain_y);	
	*/
	__syncthreads();
	
    // Read cell
    // int myself = read_cell(sdata, tx, ty, 0, 0, domain_x, 3);
    
    // Read the 8 neighbors and count number of blue and red
	int i;
	int count_blue = 0, count_red = 0, valTemp;
	
	// parcours des voisins
	for (i=-1; i<2; i++){
	
		valTemp = read_cell(sdata, tx, ty, i, -1, sdataDim, sdataDim);
		switch(valTemp){
			case 1:	
				count_red++;
				break;
			case 2:
				count_blue++;
				break;
		}
		
		valTemp = read_cell(sdata, tx, ty, i, 1, sdataDim, sdataDim);
		switch(valTemp){
			case 1:	
				count_red++;
				break;
			case 2:
				count_blue++;
				break;
		}
	}
	
	valTemp = read_cell(sdata, tx, ty, -1, 0, sdataDim, sdataDim);
	switch(valTemp){
		case 1:	
			count_red++;
			break;
		case 2:
			count_blue++;
			break;
	}
		
	valTemp = read_cell(sdata, tx, ty, 1, 0, sdataDim, sdataDim);
	switch(valTemp){
		case 1:	
			count_red++;
			break;
		case 2:
			count_blue++;
			break;
	}
	
	// Compute new value
	int new_cell=0;
	int num_nei = count_red + count_blue;
	
	switch (myself){
	
	case 0: // empty cell 
		if (num_nei == 3) { // neighbors == 3
			if (count_red < count_blue)	new_cell = 2;
			else new_cell = 1;
		}
		break;
	default: // cell survives if neighbors == 2|3
		if (num_nei == 2 || num_nei == 3){ new_cell = myself;}
		break;
	}
	
	// Write it in dest_domain	
	// sync et recopie
	 __syncthreads();

	dest_domain[ty * domain_x + tx] = new_cell;
}	

