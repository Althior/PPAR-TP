
#include <hip/hip_runtime.h>
__global__ void init_kernel(int * domain, int domain_x)
{
	// Dummy initialization
	domain[blockIdx.y * domain_x + blockIdx.x * blockDim.x + threadIdx.x]
		= (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain,
    int domain_x, int domain_y)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y;
    
	// Copier depuis la mémoire globale vers la mémoire partagée pour que les threads y accèdent par la suite
	// sdata -> Bloc = 8*8 threads -> 100 lectures (10*10) -> |sdata| = 100
	extern __shared__ int sdata[];
	int sdataDim = 10; // Voir si on ne peut pas obtenir ça de manière auto
	
	// Chaque thread lit sa case
	int myself = read_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y);
	
	/*
	*	Dans le cas d'un bloc 3x3:
	*	bloc: x² (carré pour optimiser lectures)
	*	 0 1 2
	*	 3 4 5
	*	 6 7 8
	*
	*	sdata: (x+2)² (sdataDim=5)
	*	 0  1  2  3  4
	*	 5| 6  7  8| 9
	*	10|11 12 13|14
	*	15|16 17 18|19
	*	20 21 22 23 24
	*/
	
	// Position y du thread dans la shared memory
	int decY, myloc;
	decY = (threadIdx.x / blockDim.x) + 1;
	//decY = (threadIdx.x / sdataDim) + 1; // Mieux ?
	
	// Position dans sdata (avec contours compris)
	myloc = sdataDim*decY + 1 + (threadIdx.x % blockDim.x); // +1 nécessaire ?
	sdata[myloc] = myself;
	
	/*
		Initialisation de 4 variables par thread : haut bas gauche droite
		Booléens indiquant si le thread est situé en bordure du carré de cellules traité
	*/
	int haut, bas, gauche, droite;
	haut   = threadIdx.x < blockDim.x;
	bas    = threadIdx.x >= blockDim.x * (blockDim.y -1);
	gauche = (threadIdx.x % blockDim.x) == 0;
	droite = ((threadIdx.x + 1) % blockDim.x) == 0;
	
	/* Lectures des bordures si nécessaire */
	
	// Lecture en haut
	if (haut) {
	
		sdata[myloc-sdataDim] = read_cell(source_domain, tx, ty, 0, -1, domain_x, domain_y);
		
		// Lecture haut-gauche
		if (gauche) {
			
			sdata[myloc-sdataDim-1] = read_cell(source_domain, tx, ty, -1, -1, domain_x, domain_y);
		}
		
		// Lecture haut-droite
		if (droite) {
			
			sdata[myloc-sdataDim+1] = read_cell(source_domain, tx, ty, 1, -1, domain_x, domain_y);
		}
	}
	
	// Lecture en bas
	if (bas) {
		
		sdata[myloc+sdataDim] = read_cell(source_domain, tx, ty, 0, 1, domain_x, domain_y);
		
		// Lecture bas-gauche
		if (gauche) {
			
			sdata[myloc+sdataDim-1] = read_cell(source_domain, tx, ty, -1, 1, domain_x, domain_y);
		}
		
		// Lecture bas-droite
		if (droite) {
			
			sdata[myloc+sdataDim+1] = read_cell(source_domain, tx, ty, 1, 1, domain_x, domain_y);
		}
	}
	
	// Lecture à gauche
	if (gauche) {
	
		sdata[myloc-1] = read_cell(source_domain, tx, ty, -1, 0, domain_x, domain_y);
	}
	
	// Lecture à droite
	if (droite) {
		
		sdata[myloc+1] = read_cell(source_domain, tx, ty, 1, 0, domain_x, domain_y);
	}	

	/*
	sdata[threadIdx.x] = read_cell(source_domain, tx, ty, 0, -1, domain_x, domain_y);
	sdata[threadIdx.x+blockDim.x] = read_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y);
	sdata[threadIdx.x+(2*blockDim.x)] = read_cell(source_domain, tx, ty, 0, 1, domain_x, domain_y);	
	*/
	__syncthreads();
	
    // Read cell
    // int myself = read_cell(sdata, tx, ty, 0, 0, domain_x, 3);
    
    // Read the 8 neighbors and count number of blue and red
	int i;
	int count_blue = 0, count_red = 0, valTemp;
	
	// Parcours des voisins du dessus et du dessous
	for (i=-1; i<2; i++){
	
		valTemp = read_cell(sdata, tx, ty, i, -1, sdataDim, sdataDim);
		switch(valTemp){
			case 1:	
				count_red++;
				break;
			case 2:
				count_blue++;
				break;
		}
		
		valTemp = read_cell(sdata, tx, ty, i, 1, sdataDim, sdataDim);
		switch(valTemp){
			case 1:	
				count_red++;
				break;
			case 2:
				count_blue++;
				break;
		}
	}
	
	// Voisin gauche
	valTemp = read_cell(sdata, tx, ty, -1, 0, sdataDim, sdataDim);
	switch(valTemp){
		case 1:	
			count_red++;
			break;
		case 2:
			count_blue++;
			break;
	}
	
	// Voisin droit	
	valTemp = read_cell(sdata, tx, ty, 1, 0, sdataDim, sdataDim);
	switch(valTemp){
		case 1:	
			count_red++;
			break;
		case 2:
			count_blue++;
			break;
	}
	
	// Compute new value
	int new_cell=0;
	int num_nei = count_red + count_blue;
	
	switch (myself){
	
	case 0: // Cellule vide
		if (num_nei == 3) { // 3 voisins
			if (count_red < count_blue)	new_cell = 2;
			else new_cell = 1;
		}
		break;
	default: // Survie de la cellule si nbVoisins == 2||3
		if (num_nei == 2 || num_nei == 3){ new_cell = myself;}
		break;
	}
	
	// Write it in dest_domain	
	// sync et recopie
	 __syncthreads();

	dest_domain[ty * domain_x + tx] = new_cell;
}	

